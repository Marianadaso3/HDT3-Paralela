
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    int threadID = threadIdx.x; // Modifica la forma en que se calcula el identificador del hilo
    printf("Hello, World! I'm thread %d. Nuestro nombres son Mariana David 201055, Angel Higueros 20460 y Pablo Escobar 20936\n", threadID);
    
    // if (threadID == 0) {
    //     printf("Kernel ejecutado correctamente\n");
    // } else {
    //     printf("Error en el kernel: threadID != 0\n");
    // }
}

int main()
{
    int numBlocks = 1;          // Número de bloques (1 bloque)
    int threadsPerBlock = 1024; // Número de hilos por bloque (2048 hilos)

    // Llama al kernel con la configuración de bloques y hilos
    helloCUDA<<<numBlocks, threadsPerBlock>>>();
    helloCUDA<<<numBlocks, threadsPerBlock>>>();

    // Espera a que todos los hilos hayan terminado
    hipDeviceSynchronize();
    return 0;
}